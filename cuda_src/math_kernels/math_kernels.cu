#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : math_kernels.cu
 Author      : Shuyang Sun
 Version     :
 Copyright   : Shuyang Sun, all rights reserved.
 ============================================================================
 */

#include "math_kernels.hpp"

__global__ void FPO_1() {
  float const res{1.0f * 1.5f};
}
__global__ void FPO_2() {
  float const res{1.0f * 1.5f + 2.0f};
}
__global__ void FPO_3() {
  float const res{1.0f * 1.5f + 2.0f * 8.0f};
}

__global__ void FPO_4() {
  float const res{1.0f * 1.5f + 2.0f * 8.0f - 9.6f};
}

__global__ void FPO_5() {
  float const res{1.0f * 1.5f + 2.0f * 8.0f - 9.6f * 2.0f};
}

__global__ void FPO_6() {
  float const res{1.0f * 1.5f + 2.0f * 8.0f - 9.6f * 2.0f + 3.0f};
}

__global__ void FPO_7() {
  float const res{1.0f * 1.5f + 2.0f * 8.0f - 9.6f * 2.0f + 3.0f - 2.5f};
}

__global__ void FPO_8() {
  float const res{1.0f * 1.5f + 2.0f * 8.0f - 9.6f * 2.0f + 3.0f - 2.5f * 0.5f};
}

__global__ void FPO_9() {
  float const res{1.0f * 1.5f + 2.0f * 8.0f - 9.6f * 2.0f + 3.0f - 2.5f * 0.5f - 2.4f};
}

__global__ void FPO_10() {
  float const res{1.0f * 1.5f + 2.0f * 8.0f - 9.6f * 2.0f + 3.0f - 2.5f * 0.5f - 2.4f + 2.3f};
}

__global__ void FPO_11() {
  float const res{1.0f * 1.5f + 2.0f * 8.0f - 9.6f * 2.0f + 3.0f - 2.5f * 0.5f - 2.4f + 2.3f * 8.9f};
}

__global__ void FPO_12() {
  float const res{1.0f * 1.5f + 2.0f * 8.0f - 9.6f * 2.0f + 3.0f - 2.5f * 0.5f - 2.4f + 2.3f * 8.9f - 3.5f};
}

__global__ void FPO_13() {
  float const res{1.0f * 1.5f + 2.0f * 8.0f - 9.6f * 2.0f + 3.0f - 2.5f * 0.5f - 2.4f + 2.3f * 8.9f - 3.5f + 11.6f};
}

__global__ void FPO_14() {
  float const res{1.0f * 1.5f + 2.0f * 8.0f - 9.6f * 2.0f + 3.0f - 2.5f * 0.5f - 2.4f + 2.3f * 8.9f - 3.5f + 11.6f - 7.8f};
}

__global__ void FPO_15() {
  float const res{1.0f * 1.5f + 2.0f * 8.0f - 9.6f * 2.0f + 3.0f - 2.5f * 0.5f - 2.4f + 2.3f * 8.9f - 3.5f + 11.6f - 7.8f * 3.0f};
}

__global__ void FPO_16() {
  float const res{1.0f * 1.5f + 2.0f * 8.0f - 9.6f * 2.0f + 3.0f - 2.5f * 0.5f - 2.4f + 2.3f * 8.9f - 3.5f + 11.6f - 7.8f * 3.0f - 2.0f};
}


__global__ void SFU_1() {
  float const res{__sinf(2.0f)};
}

__global__ void SFU_2() {
  float const res{__sinf(__cosf(2.0f))};
}

__global__ void SFU_3() {
  float const res{__sinf(__cosf(__sinf(2.0f)))};
}

__global__ void SFU_4() {
  float const res{__sinf(__cosf(__sinf(__cosf(2.0f))))};
}

__global__ void SFU_5() {
  float const res{__sinf(__cosf(__sinf(__cosf(__sinf(2.0f)))))};
}

__global__ void SFU_6() {
  float const res{__sinf(__cosf(__sinf(__cosf(__sinf(__cosf(2.0f))))))};
}

__global__ void SFU_7() {
  float const res{__sinf(__cosf(__sinf(__cosf(__sinf(__cosf(__sinf(2.0f)))))))};
}

__global__ void SFU_8() {
  float const res{__sinf(__cosf(__sinf(__cosf(__sinf(__cosf(__sinf(__cosf(2.0f))))))))};
}

__global__ void SFU_9() {
  float const res{__sinf(__cosf(__sinf(__cosf(__sinf(__cosf(__sinf(__cosf(__sinf(2.0f)))))))))};
}

__global__ void SFU_10() {
  float const res{__sinf(__cosf(__sinf(__cosf(__sinf(__cosf(__sinf(__cosf(__sinf(__cosf(2.0f))))))))))};
}

__global__ void SFU_11() {
  float const res{__sinf(__cosf(__sinf(__cosf(__sinf(__cosf(__sinf(__cosf(__sinf(__cosf(__sinf(2.0f)))))))))))};
}

__global__ void SFU_12() {
  float const res{__sinf(__cosf(__sinf(__cosf(__sinf(__cosf(__sinf(__cosf(__sinf(__cosf(__sinf(__cosf(2.0f))))))))))))};
}

__global__ void SFU_13() {
  float const res{__sinf(__cosf(__sinf(__cosf(__sinf(__cosf(__sinf(__cosf(__sinf(__cosf(__sinf(__cosf(__sinf(2.0f)))))))))))))};
}

__global__ void SFU_14() {
  float const res{__sinf(__cosf(__sinf(__cosf(__sinf(__cosf(__sinf(__cosf(__sinf(__cosf(__sinf(__cosf(__sinf(__cosf(2.0f))))))))))))))};
}

__global__ void SFU_15() {
  float const res{__sinf(__cosf(__sinf(__cosf(__sinf(__cosf(__sinf(__cosf(__sinf(__cosf(__sinf(__cosf(__sinf(__cosf(__sinf(2.0f)))))))))))))))};
}

__global__ void SFU_16() {
  float const res{__sinf(__cosf(__sinf(__cosf(__sinf(__cosf(__sinf(__cosf(__sinf(__cosf(__sinf(__cosf(__sinf(__cosf(__sinf(__cosf(2.0f))))))))))))))))};
}


