#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : helper.cu
 Author      : Shuyang Sun
 Version     :
 Copyright   : Shuyang Sun, all rights reserved.
 ============================================================================
 */

#include "helper.hpp"

#include <cstdlib>
#include <iostream>

float RandomFloat() {
  return ((float)rand())/((float)rand());
}

double CPUSecond() {
  return static_cast<double>(clock()) / CLOCKS_PER_SEC;
}

void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err) {
  if (err == hipSuccess)
    return;
  std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
  exit (1);
}

