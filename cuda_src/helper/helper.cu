/*
 ============================================================================
 Name        : helper.cu
 Author      : Shuyang Sun
 Version     :
 Copyright   : Shuyang Sun, all rights reserved.
 ============================================================================
 */

#include "helper.hpp"

#include <cstdlib>
#include <iostream>

float RandomFloat() {
  return ((float)rand())/((float)rand());
}

void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err) {
  if (err == hipSuccess)
    return;
  std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
  exit (1);
}

