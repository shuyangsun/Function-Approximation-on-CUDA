/*
 ============================================================================
 Name        : main.cu
 Author      : Shuyang Sun
 Version     :
 Copyright   : Shuyang Sun, all rights reserved.
 ============================================================================
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <cstdlib>

static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CHECK_CUDA_ERR(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

double CPUSecond();
float RandomFloat();

__global__ void PolyFunc(const float * const data_in, float * const data_out, size_t const size);
__global__ void TrigFunc(const float * const data_in, float * const data_out, size_t const size);

int main(int arc, char *argv[]) {

  // Customization for testing.
  size_t const num_loop{10};
  float const max_gig_count{6.0f};
  float const step_size{1.0f};

  // Initialize attributes
  size_t const max_data_size{static_cast<size_t>(max_gig_count * 1024 * 1024 * 1024)};
  size_t const max_num_ele{max_data_size / sizeof(float)};

  // Generate random data array
  float * const data_h{reinterpret_cast<float*>(malloc(max_data_size))};
  std::cout << std::setprecision(2) << "Generating random float array of " << max_gig_count << "GB..." << std::endl;
  srand(time(NULL));
  for (size_t i{0}; i < max_num_ele; ++i) {
    data_h[i] = RandomFloat();
  }
  std::cout << "Finished generating random float array." << std::endl;

  // Start outter loop (data size loop)
  for (float i{step_size}; i <= max_gig_count; i += step_size) {
    std::cout << std::setprecision(2) << "------------ " << i << "GB ------------" << std::endl;
    double duration_trig{0.0};
    double duration_poly{0.0};

    // Start inner loop (repetition loop)
    for (size_t j{0}; j < num_loop; ++j) {

      float const gig_count{i};
      size_t const data_size{static_cast<size_t>(gig_count * (1 << 30))};
      size_t const num_ele{data_size / sizeof(float)};

      float *data_d;
      CHECK_CUDA_ERR(hipMalloc(reinterpret_cast<void**>(&data_d), data_size));
      CHECK_CUDA_ERR(hipMemcpy(data_d, data_h, data_size, hipMemcpyHostToDevice));

      float *res;
      CHECK_CUDA_ERR(hipMalloc(reinterpret_cast<void**>(&res), data_size/2));

      const dim3 block_dim{1024};
      const dim3 grid_dim{static_cast<unsigned int>((num_ele + block_dim.x - 1) / block_dim.x)};

      // Trig Kernel
      double start{CPUSecond()};
      TrigFunc<<<grid_dim, block_dim>>>(data_d, res, num_ele);
      hipDeviceSynchronize();
      double end{CPUSecond()};
      duration_trig += (end - start) * 1000.0;

      CHECK_CUDA_ERR(hipFree(res));
      CHECK_CUDA_ERR(hipMalloc(reinterpret_cast<void**>(&res), data_size/2));

      // Poly Kernel
      start = CPUSecond();
      PolyFunc<<<grid_dim, block_dim>>>(data_d, res, num_ele);
      hipDeviceSynchronize();
      end = CPUSecond();
      duration_poly += (end - start) * 1000.0;

      CHECK_CUDA_ERR(hipFree(res));
      CHECK_CUDA_ERR(hipFree(data_d));

    }

    // Calculate average
    duration_trig /= num_loop;
    duration_poly /= num_loop;

    // Print out information
    std::cout << std::setprecision(3) << "Finished trig kernel in average " << duration_trig << " ms." << std::endl;
    std::cout << std::setprecision(3) << "Finished poly kernel in average " << duration_poly << " ms." << std::endl;
    std::cout << std::setprecision(5) << "Trig time / Poly time: " << duration_trig/duration_poly << std::endl;
    std::cout << std::setprecision(3) << "Speed up: " << (1.0f - duration_poly/duration_trig) * 100 << "%" << std::endl;
  }

  free(data_h);
  CHECK_CUDA_ERR(hipDeviceReset());

  std::cout << "-------------------------------" << std::endl;
  return 0;
}

__global__ void PolyFunc(const float * const data_in, float * const data_out, size_t const size) {

  const size_t idx{threadIdx.x + blockIdx.x * blockDim.x};
  const size_t idx_2{idx * 2};

  if (idx_2 < size) {
    const float a{data_in[idx_2]};
    const float b{data_in[idx_2 + 1]};
    const float res1 = 0.36f + 0.68f * a * (1 + 0.28f * a * (1 + 0.78f * a * (1 - 0.57f * a * (1 + 0.68f * a * (1 + 0.68f * a * (1 + 0.68f * a * (1 + 0.68f * a)))))));
    const float res2 = 0.36f + 0.68f * b * (1 + 0.28f * b * (1 + 0.78f * b * (1 - 0.57f * b * (1 + 0.68f * b * (1 + 0.68f * b * (1 + 0.68f * b * (1 + 0.68f * b)))))));

    data_out[idx] = res2 - res1;
  }
}

__global__ void TrigFunc(const float * const data_in, float * const data_out, size_t const size) {
  const size_t idx{threadIdx.x + blockIdx.x * blockDim.x};
  const size_t idx_2{idx * 2};
  if (idx_2 < size) {
    const float a = data_in[idx_2];
    const float b = data_in[idx_2 + 1];
    const float res1 = 2.5f * __sinf(a) + 0.25f * __sinf(3 * a) - 8.5f * __cosf(a) + 9.3f * __cosf(3 * a) + 0.34f * __cosf(5 * a) + 9.3f * __cosf(7 * a);
    const float res2 = 2.5f * __sinf(b) + 0.25f * __sinf(3 * b) - 8.5f * __cosf(b) + 9.3f * __cosf(3 * b) + 0.34f * __cosf(5 * b) + 9.3f * __cosf(7 * b);

    data_out[idx] = res2 - res1;
  }
}

float RandomFloat() {
  return ((float)rand())/((float)rand());
}

double CPUSecond() {
  return static_cast<double>(clock()) / CLOCKS_PER_SEC;
}

static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err) {
  if (err == hipSuccess)
    return;
  std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
  exit (1);
}

